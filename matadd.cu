#include "hip/hip_runtime.h"
% % cu
#include <stdio.h>
#include <hip/hip_runtime.h>

        __global__ void
        add(int *d, int *e, int *f)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int id = gridDim.x * y + x;
    f[id] = d[id] + e[id];
}
int main()
{
    int a[2][3] = {{1, 2, 3}, {4, 5, 6}}, b[2][3] = {{1, 2, 3}, {4, 5, 6}}, c[2][3], *d, *e, *f;
    hipMalloc((void **)&d, 6 * sizeof(int));
    hipMalloc((void **)&e, 6 * sizeof(int));
    hipMalloc((void **)&f, 6 * sizeof(int));
    hipMemcpy(d, &a, 6 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(e, &b, 6 * sizeof(int), hipMemcpyHostToDevice);
    dim3 grid(3, 2);
    add<<<grid, 1>>>(d, e, f);
    hipMemcpy(&c, f, 6 * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 2; i++)
        for (int j = 0; j < 2; j++)
            printf("%d\t", c[i][j]);
    printf("\n");
    hipFree(d);
    hipFree(e);
    hipFree(f);
    hipDeviceSynchronize();
    return 0;
}